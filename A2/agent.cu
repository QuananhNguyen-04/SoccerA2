#include <iostream>
#include <vector>
#include <cmath>
#include <random>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void initializeWeightsKernel(double* weights, int fan_in, int fan_out, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < fan_in * fan_out) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        weights[idx] = hiprand_normal(&state) * sqrt(2.0 / fan_in);
    }
}

class NeuralNetwork {
public:
    NeuralNetwork(const std::vector<int>& layers);
    ~NeuralNetwork();
    std::vector<std::vector<double>> forward(const std::vector<std::vector<double>>& inputs);
    void backward(const std::vector<std::vector<double>>& inputs, const std::vector<std::vector<double>>& gradients);
    void saveNetwork(const std::string& filename) const;
    void loadNetwork(const std::string& filename);

private:
    void initializeWeights();
    void initializeBiases();
    void initializeAdam();
    double sigmoid(double x) const;
    double sigmoidDerivative(double x) const;

    std::vector<int> layers;
    std::vector<std::vector<std::vector<double>>> weights;
    std::vector<std::vector<double>> biases;
    std::vector<std::vector<std::vector<double>>> m;
    std::vector<std::vector<std::vector<double>>> v;
    std::vector<std::vector<std::vector<double>>> activations;
    double beta1, beta2, epsilon, learning_rate;
    int t;
};

NeuralNetwork::NeuralNetwork(const std::vector<int>& layers)
    : beta1(0.9), beta2(0.999), epsilon(1e-10), learning_rate(0.001), t(0) {
    if (layers.size() < 2)
        throw std::invalid_argument("Network must have at least one input and one output layer.");

    this->layers = layers;
    initializeWeights();
    initializeBiases();
    initializeAdam();
}

NeuralNetwork::~NeuralNetwork() {
    // Free CUDA memory if allocated
}

void NeuralNetwork::initializeWeights() {
    weights.clear();
    for (size_t i = 0; i < layers.size() - 1; ++i) {
        int fan_in = layers[i];
        int fan_out = layers[i + 1];
        std::vector<std::vector<double>> layerWeights(fan_out, std::vector<double>(fan_in));

        double* d_weights;
        hipMalloc(&d_weights, fan_in * fan_out * sizeof(double));
        initializeWeightsKernel<<<(fan_in * fan_out + 255) / 256, 256>>>(d_weights, fan_in, fan_out, time(0));
        hipMemcpy(layerWeights.data(), d_weights, fan_in * fan_out * sizeof(double), hipMemcpyDeviceToHost);
        hipFree(d_weights);

        weights.push_back(layerWeights);
    }
}

void NeuralNetwork::initializeBiases() {
    biases.clear();
    for (size_t i = 1; i < layers.size(); ++i) {
        biases.push_back(std::vector<double>(layers[i], 0.0));
    }
}

void NeuralNetwork::initializeAdam() {
    m.clear();
    v.clear();
    for (size_t i = 0; i < weights.size(); ++i) {
        std::vector<std::vector<double>> layerM(layers[i + 1], std::vector<double>(layers[i], 0.0));
        std::vector<std::vector<double>> layerV(layers[i + 1], std::vector<double>(layers[i], 0.0));
        m.push_back(layerM);
        v.push_back(layerV);
    }
}

std::vector<std::vector<double>> NeuralNetwork::forward(const std::vector<std::vector<double>>& inputs) {
    // ...existing code...
}

void NeuralNetwork::backward(const std::vector<std::vector<double>>& inputs, const std::vector<std::vector<double>>& gradients) {
    // ...existing code...
}

void NeuralNetwork::saveNetwork(const std::string& filename) const {
    // ...existing code...
}

void NeuralNetwork::loadNetwork(const std::string& filename) {
    // ...existing code...
}

double NeuralNetwork::sigmoid(double x) const {
    return 1.0 / (1.0 + exp(-x));
}

double NeuralNetwork::sigmoidDerivative(double x) const {
    double sig = sigmoid(x);
    return sig * (1 - sig);
}

class Agent {
public:
    Agent(const std::vector<int>& layers, const std::vector<int>& vlayers, double gamma, double lambda);
    std::vector<int> takeAction(const std::vector<double>& input, bool on_policy);
    double computeAdvantage(const std::vector<double>& rewards, const std::vector<double>& values, const std::vector<double>& next_values, const std::vector<bool>& dones);
    void update(const std::vector<std::vector<double>>& states, const std::vector<double>& rewards, const std::vector<std::vector<double>>& next_states, const std::vector<bool>& dones);
    void saveWeights(const std::string& filename);
    void loadWeights(const std::string& filename);

private:
    NeuralNetwork policyNetwork;
    NeuralNetwork valueNetwork;
    double gamma, lambda;
    int time_step;
    double epsilon;
};

Agent::Agent(const std::vector<int>& layers, const std::vector<int>& vlayers, double gamma, double lambda)
    : policyNetwork(layers), valueNetwork(vlayers), gamma(gamma), lambda(lambda), time_step(0), epsilon(0.999) {
}

std::vector<int> Agent::takeAction(const std::vector<double>& input, bool on_policy) {
    // ...existing code...
}

double Agent::computeAdvantage(const std::vector<double>& rewards, const std::vector<double>& values, const std::vector<double>& next_values, const std::vector<bool>& dones) {
    // ...existing code...
}

void Agent::update(const std::vector<std::vector<double>>& states, const std::vector<double>& rewards, const std::vector<std::vector<double>>& next_states, const std::vector<bool>& dones) {
    // ...existing code...
}

void Agent::saveWeights(const std::string& filename) {
    policyNetwork.saveNetwork(filename + "_policy");
    valueNetwork.saveNetwork(filename + "_value");
}

void Agent::loadWeights(const std::string& filename) {
    policyNetwork.loadNetwork(filename + "_policy");
    valueNetwork.loadNetwork(filename + "_value");
}
